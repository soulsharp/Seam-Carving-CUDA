
#include <hip/hip_runtime.h>
# include <cmath>
# include <cfloat>

extern "C" __global__ void Rgb2GrayWithPadding(unsigned char* img, unsigned char* grayImg, int width, int height){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Padding = 1
    int paddedWidth = width + 2;
    int paddedHeight = height + 2;

    // Checks if the thread accesses an out of bounds index
    if(x >= paddedWidth || y >= paddedHeight){
        return;
    }

    // Sets the padding region to 0
    if(x == 0 || x == width + 1 || y == 0 || y == height + 1){
        grayImg[y * paddedWidth + x] = 0;
        return;
    }

    // Maps to a section of the original image
    int idx = ((y-1) * width + (x - 1)) * 3;
    unsigned char red = img[idx];
    unsigned char green = img[idx + 1];
    unsigned char blue = img[idx + 2];

    // calculates grayscale value based on RGB values for a pixel
    float grayVal = 0.2125f * red + 0.7154f * green + 0.0721f * blue;

    // Writes the grayscale value back
    grayImg[y * paddedWidth + x] = static_cast<unsigned char>(grayVal);
}

// This kernel applies the sobel filter on a grayscale image to detect gradients in the X-direction
extern "C" __global__ void SobelHorizontal(unsigned char* grayImg, float* sobelX, int width, int height) {

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;

    // Accessing corresponding position in padded image
    int paddedX = x + 1;
    int paddedY = y + 1;
    int paddedWidth = width + 2;

    // Accessing mid idx of every row involved in the Sobel operation
    int grayTopIdx = (paddedY - 1) * paddedWidth + paddedX;
    int grayLowIdx = (paddedY + 1) * paddedWidth + paddedX;

    // Sobel calculation
    float topVal = -grayImg[grayTopIdx - 1] - 2 * grayImg[grayTopIdx] - grayImg[grayTopIdx + 1];
    float lowVal =  grayImg[grayLowIdx - 1] + 2 * grayImg[grayLowIdx] + grayImg[grayLowIdx + 1];
    float sobelVal = fabsf(topVal + lowVal);

    // Writes to sobelX map
    sobelX[idx] = sobelVal;
}

// This kernel applies the sobel filter on a grayscale image to detect gradients in the Y-direction
extern "C" __global__ void SobelVertical(unsigned char* grayImg, float* sobelY, int width, int height) {

    // sobel_y -> height x width
    // grayImg -> (height + 2) x (width + 2)

    // sobel_y - > [[-1, 0, -1], [-2, 0, 2], [-1, 0, 1]]

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= width || y >= height) return;

    // Accessing corresponding position in padded image
    int paddedX = x + 1;
    int paddedY = y + 1;

    // Width of the grayImg
    int paddedWidth = width + 2;

    // Index into Sobel_Y
    int idx = y * width + x;

    // Accessing mid idx of every row involved in the Sobel operation
    int grayTopIdx = (paddedY - 1) * paddedWidth + paddedX;
    int grayMidIdx = paddedY * paddedWidth + paddedX;
    int grayLowIdx = (paddedY + 1) * paddedWidth + paddedX;
    
    // The contribution of the middle column is 0 but is included for readability
    float topVal = -grayImg[grayTopIdx - 1] + 0 + grayImg[grayTopIdx + 1];
    float mid_val = -2 * grayImg[grayMidIdx - 1] + 0 + 2 * grayImg[grayMidIdx + 1];
    float lowVal = -grayImg[grayLowIdx - 1] + 0 + grayImg[grayLowIdx + 1];

    // Calculates sobelVal for idx
    sobelY[idx] = fabsf(topVal + mid_val + lowVal);
}

extern "C" __global__ void EnergyMapBackward(float* sobelX, float* sobelY, float* energyMap, int width, int height){
    // EnergyMap is just an elementwise sum of sobelX and sobel_y
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    // Checks if the thread accesses an out of bounds index
    if(x >= width || y >= height) return;

    int idx = y * width + x;

    // Calculates EnergyValue corresponding to idx
    float energyVal = sobelX[idx] + sobelY[idx];
    energyMap[idx] = energyVal;
}

extern "C" __global__ void findMinInThreadBlock(float* inputRow, int* minIndices, int length) {
    __shared__ float minValuesShared[1024];
    __shared__ int minIndicesShared[1024];

    int tid = threadIdx.x;
    int globalTid = tid + blockDim.x * blockIdx.x;

    // Loads elements into shared memory
    if (globalTid < length) {
        minValuesShared[tid] = inputRow[globalTid];
        minIndicesShared[tid] = globalTid;
    } else {
        minValuesShared[tid] = FLT_MAX;
        minIndicesShared[tid] = -1;
    }
    __syncthreads();

    // Parallel reduction to find min value and corresponding index
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            int right = tid + s;
            if (right < blockDim.x && minValuesShared[right] < minValuesShared[tid]) {
                minValuesShared[tid] = minValuesShared[right];
                minIndicesShared[tid] = minIndicesShared[right];
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        minIndices[blockIdx.x] = minIndicesShared[0];
    }
}

extern "C" __global__ void cumulativeMapBackward(float* energyMap, float* cumulativeEnergyMap,
                                int imageHeight, int imageWidth){

    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // Checks if the thread accesses an out of bounds index
    if(tid >= 1024) return;

    // Starts computation from row no 2
    for(int rowIdx = 1; rowIdx < imageHeight; ++rowIdx){
      for(int pixelIdx = tid; pixelIdx < imageWidth; pixelIdx += 1024){
        int prevRowStartIdx = (rowIdx - 1) * imageWidth;
        int currentRowStartIdx = rowIdx * imageWidth;
        int elementAbove = prevRowStartIdx + pixelIdx;
        int currentElement = currentRowStartIdx + pixelIdx;
        int energyToAdd = 0.0;

        // At the leftmost position
        if(pixelIdx % imageWidth == 0){
            energyToAdd = fminf(cumulativeEnergyMap[elementAbove], cumulativeEnergyMap[elementAbove + 1]);
        }

        // At the rightmost position
        else if((pixelIdx + 1) % imageWidth == 0){
            energyToAdd = fminf(cumulativeEnergyMap[elementAbove], cumulativeEnergyMap[elementAbove - 1]);
        }

        // Remaining positions
        else{
            int temp = fminf(cumulativeEnergyMap[elementAbove], cumulativeEnergyMap[elementAbove + 1]);
            energyToAdd = fminf(temp, cumulativeEnergyMap[elementAbove - 1]);
        }

        cumulativeEnergyMap[currentElement] = energyMap[currentElement] + energyToAdd;
     }
     __syncthreads();
    }
}

extern "C" __global__ void removeVerticalSeam(int* seamIndices, unsigned char* gray, unsigned char* grayNew,
                                            int energyMapWidth, int energyMapHeight){
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Threads also handle the padding region
    if (tid >= ((energyMapHeight + 2) * (energyMapWidth + 1))) return;
    
    // Row, Col handled by the present thread
    int rowIdx = tid / (energyMapWidth + 1);
    int colIdx = tid % (energyMapWidth + 1);
    
    int seamCol = seamIndices[rowIdx];

    // Amount by which pixels from the old image have to be shifted in the new image
    int amountLeftShift; 
    if (colIdx < seamCol){
        amountLeftShift = rowIdx;
    }
    else{
        amountLeftShift = rowIdx + 1;
    }

    grayNew[tid] = gray[tid + amountLeftShift];
} 


extern "C" __global__ void removeVerticalSeamAndInsertPadding(int* seamIndices, unsigned char* gray, unsigned char* grayNew,
                                                             int energyMapWidth, int energyMapHeight) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Bounds check
    if (x >= energyMapWidth + 1 || y >= energyMapHeight + 2) {
        return;
    }

    // Index in grayNew where this thread writes
    int grayNewIdx = y * (energyMapWidth + 1) + x;

    // Sets padding outside the actual content region to 0
    if (x == 0 || x == energyMapWidth || y == 0 || y == energyMapHeight + 1) {
        grayNew[grayNewIdx] = 0;
        return;
    }

    // Gets seam index for the current row (adjusted for padding)
    int k = seamIndices[y - 1];

    // Computes the corresponding index in gray
    int grayOldIdx = y * (energyMapWidth + 2) + x;

    // Pixels before the seam pixel remain the same, pixels after get shifted to the left by 1
    if (x < k) {
        grayNew[grayNewIdx] = gray[grayOldIdx];
    }
    else {
        grayNew[grayNewIdx] = gray[grayOldIdx + 1];

    }
}

extern "C" __global__ void removeSeamRGB(unsigned char* red, unsigned char* green, unsigned char* blue,
                                        unsigned char* redNew, unsigned char* greenNew, unsigned char* blueNew, 
                                        int* seamIndices, int width, int height) {

    int y = threadIdx.y + blockIdx.y * blockDim.y; 
    int x = threadIdx.x + blockIdx.x * blockDim.x;  

    // Bounds check
    if (y >= height || x >= width - 1) return;

    int k = seamIndices[y];  
    if (x < k) return;       

    // Shift pixels left
    int idx = y * width + x;
    redNew[idx] = red[idx + 1];
    greenNew[idx] = green[idx + 1];
    blueNew[idx] = blue[idx + 1];
}

extern "C" __global__ void updateEnergyMap(int* seamIndices, unsigned char* grayImg, 
                                         float* sobelX, float* sobelY, float* energyMap, 
                                         int width, int height) {
    // x -> [-1, 0, 1] 
    // y -> [0, height - 1]

    // -1 to shift the range to [-1, 0, 1]
    int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Bounds check (ensures within valid row indices)
    if (y >= height) return;

    // Computes the column index of the affected pixel
    int seamIdx = seamIndices[y]; 
    int updatedCol = seamIdx + x; 

    // Ensures the updated pixel is within bounds
    if (updatedCol < 1 || updatedCol >= width - 1) return;

    // Converts to padded image coordinates
    int paddedY = y + 1; 
    int paddedWidth = width + 2;
    int paddedIdx = paddedY * paddedWidth + updatedCol;

    // Gets indices for the 3x3 neighborhood needed for Sobel filter
    int topIdx = paddedIdx - paddedWidth;   
    int midIdx = paddedIdx;                 
    int lowIdx = paddedIdx + paddedWidth;  

    // SobelX calculation
    float sobelValX = fabsf(
    -grayImg[topIdx - 1] - 2 * grayImg[midIdx - 1] - grayImg[lowIdx - 1] +
    grayImg[topIdx + 1] + 2 * grayImg[midIdx + 1] + grayImg[lowIdx + 1]
    );

    // SobelY calculation
    float sobelValY = fabsf(
    -grayImg[topIdx - 1] - 2 * grayImg[topIdx] - grayImg[topIdx + 1] +
    grayImg[lowIdx - 1] + 2 * grayImg[lowIdx] + grayImg[lowIdx + 1]
    );

    // Computes energy
    float newEnergy = sobelValX + sobelValY;

    // Computes index in the non-padded outputs
    int outIdx = y * width + updatedCol;
    
    // Modifies maps
    sobelX[outIdx] = sobelValX;
    sobelY[outIdx] = sobelValY;
    energyMap[outIdx] = newEnergy;
}